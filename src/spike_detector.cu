#include "hip/hip_runtime.h"
/*
 *  spike_detector.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */





#include <config.h>
#include <cmath>
#include <iostream>
#include <string>
//#include <stdio.h>

#include "cuda_error.h"
#include "nestgpu.h"
#include "neuron_models.h"
#include "spike_detector.h"
				    //#include "spike_buffer.h"
//#include "parrot_neuron_variables.h"

enum {
  i_spike_detector_hold_spike_height=0,
  N_SPIKE_DETECTOR_SCAL_PARAM
};

const std::string spike_detector_scal_param_name[N_SPIKE_DETECTOR_SCAL_PARAM]
= {"hold_spike_height"};

enum {
  i_spike_detector_input_spike_height=0,
  i_spike_detector_spike_height,
  N_SPIKE_DETECTOR_SCAL_VAR
};

const std::string spike_detector_scal_var_name[N_SPIKE_DETECTOR_SCAL_VAR]
= {"input_spike_height", "spike_height"};


__global__
void spike_detector_UpdateKernel(int i_node_0, int n_node, float *var_arr,
				float *param_arr, int n_var, int n_param)
{
  int irel_node = threadIdx.x + blockIdx.x * blockDim.x;
  if (irel_node < n_node) {
    float *input_spike_height_pt = var_arr + irel_node*n_var
      + i_spike_detector_input_spike_height;
    float *spike_height_pt = var_arr + irel_node*n_var
      + i_spike_detector_spike_height;
    float *hold_spike_height_pt = param_arr + irel_node*n_param +
      i_spike_detector_hold_spike_height;
    //int i_node = i_node_0 + irel_node;
    float spike_height = *input_spike_height_pt;
    if (spike_height != 0.0) {
      if (*hold_spike_height_pt==0.0) {
	spike_height = 1.0;
      }
      *input_spike_height_pt = 0;
    }
    *spike_height_pt = spike_height;
  }
}


int spike_detector::Init(int i_node_0, int n_node, int /*n_port*/,
			int i_group, unsigned long long *seed)
{
  BaseNeuron::Init(i_node_0, n_node, 1 /*n_port*/, i_group, seed);
  node_type_ = i_spike_detector_model;

  n_scal_var_ = N_SPIKE_DETECTOR_SCAL_VAR;
  n_var_ = n_scal_var_;
  scal_var_name_ = spike_detector_scal_var_name;

  n_scal_param_ = N_SPIKE_DETECTOR_SCAL_PARAM;
  n_param_ = n_scal_param_;
  scal_param_name_ = spike_detector_scal_param_name;

  gpuErrchk(hipMalloc(&var_arr_, n_node_*n_var_*sizeof(float)));

  gpuErrchk(hipMalloc(&param_arr_, n_node_*n_param_*sizeof(float)));

  SetScalParam(0, n_node, "hold_spike_height", 1.0);

  SetScalVar(0, n_node, "input_spike_height", 0.0);

  SetScalVar(0, n_node, "spike_height", 0.0);

  // multiplication factor of input signal is always 1 for all nodes
  float input_weight = 1.0;
  gpuErrchk(hipMalloc(&port_weight_arr_, sizeof(float)));
  gpuErrchk(hipMemcpy(port_weight_arr_, &input_weight,
			 sizeof(float), hipMemcpyHostToDevice));
  port_weight_arr_step_ = 0;
  port_weight_port_step_ = 0;

  // input signal is stored in input_spike_height
  port_input_arr_ = GetVarArr() + GetScalVarIdx("input_spike_height");
  port_input_arr_step_ = n_var_;
  port_input_port_step_ = n_port_var_;

  return 0;
}

int spike_detector::Update(long long /*i_time*/, double /*t1*/)
{
  spike_detector_UpdateKernel<<<(n_node_+1023)/1024, 1024>>>
    (i_node_0_, n_node_, var_arr_, param_arr_, n_var_, n_param_);
  //gpuErrchk( hipPeekAtLastError() );
  //gpuErrchk( hipDeviceSynchronize() );

  return 0;
}

int spike_detector::Free()
{
  gpuErrchk(hipFree(var_arr_));
  gpuErrchk(hipFree(param_arr_));

  return 0;
}

spike_detector::~spike_detector()
{
  if (n_node_>0) {
    Free();
  }
}
