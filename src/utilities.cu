
#include <hip/hip_runtime.h>
/*
 *  utilities.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

int64_t
IntPow( int64_t x, unsigned int p )
{
  if ( p == 0 )
  {
    return 1;
  }
  if ( p == 1 )
  {
    return x;
  }

  int64_t tmp = IntPow( x, p / 2 );
  if ( p % 2 == 0 )
  {
    return tmp * tmp;
  }
  else
  {
    return x * tmp * tmp;
  }
}
