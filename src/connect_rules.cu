/*
 *  connect_rules.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "connect.h"
#include "connect_rules.h"
#include "distribution.h"
#include "nestgpu.h"
#include "ngpu_exception.h"
#include "remote_connect.h"
#include <config.h>
#include <iostream>

int
ConnSpec::Init()
{
  rule_ = ALL_TO_ALL;
  total_num_ = 0;
  indegree_ = 0;
  outdegree_ = 0;
  return 0;
}

ConnSpec::ConnSpec()
{
  Init();
}

int
ConnSpec::Init( int rule, int degree /*=0*/ )
{
  Init();
  if ( rule < 0 || rule > N_CONN_RULE )
  {
    throw ngpu_exception( "Unknown connection rule" );
  }
  if ( ( rule == ALL_TO_ALL || rule == ONE_TO_ONE ) && ( degree != 0 ) )
  {
    throw ngpu_exception( std::string( "Connection rule " ) + conn_rule_name[ rule ] + " does not have a degree" );
  }
  rule_ = rule;
  if ( rule == FIXED_TOTAL_NUMBER )
  {
    total_num_ = degree;
  }
  else if ( rule == FIXED_INDEGREE )
  {
    indegree_ = degree;
  }
  else if ( rule == FIXED_OUTDEGREE )
  {
    outdegree_ = degree;
  }

  return 0;
}

ConnSpec::ConnSpec( int rule, int degree /*=0*/ )
{
  Init( rule, degree );
}

int
ConnSpec::SetParam( std::string param_name, int value )
{
  if ( param_name == "rule" )
  {
    if ( value < 0 || value > N_CONN_RULE )
    {
      throw ngpu_exception( "Unknown connection rule" );
    }
    rule_ = value;
    return 0;
  }
  else if ( param_name == "indegree" )
  {
    if ( value < 0 )
    {
      throw ngpu_exception( "Indegree must be >=0" );
    }
    indegree_ = value;
    return 0;
  }
  else if ( param_name == "outdegree" )
  {
    if ( value < 0 )
    {
      throw ngpu_exception( "Outdegree must be >=0" );
    }
    outdegree_ = value;
    return 0;
  }
  else if ( param_name == "total_num" )
  {
    if ( value < 0 )
    {
      throw ngpu_exception( "total_num must be >=0" );
    }
    total_num_ = value;
    return 0;
  }

  throw ngpu_exception( "Unknown connection int parameter" );
}

bool
ConnSpec::IsParam( std::string param_name )
{
  if ( param_name == "rule" || param_name == "indegree" || param_name == "outdegree" || param_name == "total_num" )
  {
    return true;
  }
  else
  {
    return false;
  }
}

SynSpec::SynSpec()
{
  Init();
}

int
SynSpec::Init()
{
  syn_group_ = 0;
  port_ = 0;
  weight_ = 0;
  delay_ = 0;
  weight_distr_ = DISTR_TYPE_NONE;
  delay_distr_ = DISTR_TYPE_NONE;
  weight_h_array_pt_ = nullptr;
  delay_h_array_pt_ = nullptr;

  return 0;
}

SynSpec::SynSpec( float weight, float delay )
{
  Init( weight, delay );
}

int
SynSpec::Init( float weight, float delay )
{
  if ( delay < 0 )
  {
    throw ngpu_exception( "Delay must be >=0" );
  }
  Init();
  weight_ = weight;
  delay_ = delay;

  return 0;
}

SynSpec::SynSpec( int syn_group, float weight, float delay, int port /*=0*/ )
{
  Init( syn_group, weight, delay, port );
}

int
SynSpec::Init( int syn_group, float weight, float delay, int port /*=0*/ )
{
  if ( syn_group < 0 )
  { // || syn_group>n_syn_group) {
    throw ngpu_exception( "Unknown synapse group" );
  }
  if ( port < 0 )
  {
    throw ngpu_exception( "Port index must be >=0" );
  }
  Init( weight, delay );
  syn_group_ = syn_group;
  port_ = port;

  return 0;
}

int
SynSpec::SetParam( std::string param_name, int value )
{
  if ( param_name == "synapse_group" )
  {
    if ( value < 0 )
    { // || value>n_syn_group) {
      throw ngpu_exception( "Unknown synapse group" );
    }
    syn_group_ = value;
  }
  else if ( param_name == "receptor" )
  {
    if ( value < 0 )
    {
      throw ngpu_exception( "Port index must be >=0" );
    }
    port_ = value;
  }
  else if ( param_name == "weight_distribution" )
  {
    weight_distr_ = value;
    // printf("weight_distribution_ idx: %d\n", value);
  }
  else if ( param_name == "delay_distribution" )
  {
    delay_distr_ = value;
    // printf("delay_distribution_ idx: %d\n", value);
  }
  else
  {
    throw ngpu_exception( "Unknown synapse int parameter" );
  }

  return 0;
}

bool
SynSpec::IsIntParam( std::string param_name )
{
  if ( param_name == "synapse_group" || param_name == "receptor" || param_name == "weight_distribution"
    || param_name == "delay_distribution" )
  {
    return true;
  }
  else
  {
    return false;
  }
}

int
SynSpec::SetParam( std::string param_name, float value )
{
  if ( param_name == "weight" )
  {
    weight_ = value;
  }
  else if ( param_name == "delay" )
  {
    if ( value < 0 )
    {
      throw ngpu_exception( "Delay must be >=0" );
    }
    delay_ = value;
  }
  else if ( param_name == "weight_mu" )
  {
    weight_mu_ = value;
    // printf("weight_mu_: %f\n", value);
  }
  else if ( param_name == "weight_low" )
  {
    weight_low_ = value;
    // printf("weight_low_: %f\n", value);
  }
  else if ( param_name == "weight_high" )
  {
    weight_high_ = value;
    // printf("weight_high_: %f\n", value);
  }
  else if ( param_name == "weight_sigma" )
  {
    weight_sigma_ = value;
    // printf("weight_sigma_: %f\n", value);
  }
  else if ( param_name == "delay_mu" )
  {
    delay_mu_ = value;
    // printf("delay_mu_: %f\n", value);
  }
  else if ( param_name == "delay_low" )
  {
    delay_low_ = value;
    // printf("delay_low_: %f\n", value);
  }
  else if ( param_name == "delay_high" )
  {
    delay_high_ = value;
    // printf("delay_high_: %f\n", value);
  }
  else if ( param_name == "delay_sigma" )
  {
    delay_sigma_ = value;
    // printf("delay_sigma_: %f\n", value);
  }
  else
  {
    throw ngpu_exception( "Unknown synapse float parameter" );
  }
  return 0;
}

bool
SynSpec::IsFloatParam( std::string param_name )
{
  if ( param_name == "weight" || param_name == "delay" || param_name == "weight_mu" || param_name == "weight_low"
    || param_name == "weight_high" || param_name == "weight_sigma" || param_name == "delay_mu"
    || param_name == "delay_low" || param_name == "delay_high" || param_name == "delay_sigma" )
  {
    return true;
  }
  else
  {
    return false;
  }
}

int
SynSpec::SetParam( std::string param_name, float* array_pt )
{
  if ( param_name == "weight_array" )
  {
    weight_h_array_pt_ = array_pt;
    weight_distr_ = DISTR_TYPE_ARRAY;
  }
  else if ( param_name == "delay_array" )
  {
    delay_h_array_pt_ = array_pt;
    delay_distr_ = DISTR_TYPE_ARRAY;
  }
  else
  {
    throw ngpu_exception( "Unknown synapse array parameter" );
  }

  return 0;
}

bool
SynSpec::IsFloatPtParam( std::string param_name )
{
  if ( param_name == "weight_array" || param_name == "delay_array" )
  {
    return true;
  }
  else
  {
    return false;
  }
}

int
NESTGPU::Connect( inode_t i_source,
  inode_t n_source,
  inode_t i_target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );

  return conn_->connect( i_source, n_source, i_target, n_target, conn_spec, syn_spec );
}

int
NESTGPU::Connect( inode_t i_source,
  inode_t n_source,
  inode_t* target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );

  inode_t* d_target;
  CUDAMALLOCCTRL( "&d_target", &d_target, n_target * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_target, target, n_target * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  int ret = conn_->connect( i_source, n_source, d_target, n_target, conn_spec, syn_spec );
  CUDAFREECTRL( "d_target", d_target );

  return ret;
}

int
NESTGPU::Connect( inode_t* source,
  inode_t n_source,
  inode_t i_target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );

  inode_t* d_source;
  CUDAMALLOCCTRL( "&d_source", &d_source, n_source * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_source, source, n_source * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  int ret = conn_->connect( d_source, n_source, i_target, n_target, conn_spec, syn_spec );
  CUDAFREECTRL( "d_source", d_source );

  return ret;
}

int
NESTGPU::Connect( inode_t* source,
  inode_t n_source,
  inode_t* target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );
  inode_t* d_source;
  CUDAMALLOCCTRL( "&d_source", &d_source, n_source * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_source, source, n_source * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  inode_t* d_target;
  CUDAMALLOCCTRL( "&d_target", &d_target, n_target * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_target, target, n_target * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  int ret = conn_->connect( d_source, n_source, d_target, n_target, conn_spec, syn_spec );
  CUDAFREECTRL( "d_source", d_source );
  CUDAFREECTRL( "d_target", d_target );

  return ret;
}

int
NESTGPU::Connect( NodeSeq source, NodeSeq target, ConnSpec& conn_spec, SynSpec& syn_spec )
{
  return Connect( source.i0, source.n, target.i0, target.n, conn_spec, syn_spec );
}

int
NESTGPU::Connect( NodeSeq source, std::vector< inode_t > target, ConnSpec& conn_spec, SynSpec& syn_spec )
{
  return Connect( source.i0, source.n, target.data(), target.size(), conn_spec, syn_spec );
}

int
NESTGPU::Connect( std::vector< inode_t > source, NodeSeq target, ConnSpec& conn_spec, SynSpec& syn_spec )
{
  return Connect( source.data(), source.size(), target.i0, target.n, conn_spec, syn_spec );
}

int
NESTGPU::Connect( std::vector< inode_t > source, std::vector< inode_t > target, ConnSpec& conn_spec, SynSpec& syn_spec )
{
  return Connect( source.data(), source.size(), target.data(), target.size(), conn_spec, syn_spec );
}

int
NESTGPU::RemoteConnect( int i_source_host,
  inode_t i_source,
  inode_t n_source,
  int i_target_host,
  inode_t i_target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );

  return conn_->remoteConnect(
    i_source_host, i_source, n_source, i_target_host, i_target, n_target, conn_spec, syn_spec );
}

int
NESTGPU::RemoteConnect( int i_source_host,
  inode_t i_source,
  inode_t n_source,
  int i_target_host,
  inode_t* target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );

  inode_t* d_target;
  CUDAMALLOCCTRL( "&d_target", &d_target, n_target * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_target, target, n_target * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  int ret =
    conn_->remoteConnect( i_source_host, i_source, n_source, i_target_host, d_target, n_target, conn_spec, syn_spec );
  CUDAFREECTRL( "d_target", d_target );

  return ret;
}

int
NESTGPU::RemoteConnect( int i_source_host,
  inode_t* source,
  inode_t n_source,
  int i_target_host,
  inode_t i_target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );

  inode_t* d_source;
  CUDAMALLOCCTRL( "&d_source", &d_source, n_source * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_source, source, n_source * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  int ret =
    conn_->remoteConnect( i_source_host, d_source, n_source, i_target_host, i_target, n_target, conn_spec, syn_spec );
  CUDAFREECTRL( "d_source", d_source );

  return ret;
}

int
NESTGPU::RemoteConnect( int i_source_host,
  inode_t* source,
  inode_t n_source,
  int i_target_host,
  inode_t* target,
  inode_t n_target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  CheckUncalibrated( "Connections cannot be created after calibration" );

  inode_t* d_source;
  CUDAMALLOCCTRL( "&d_source", &d_source, n_source * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_source, source, n_source * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  inode_t* d_target;
  CUDAMALLOCCTRL( "&d_target", &d_target, n_target * sizeof( inode_t ) );
  gpuErrchk( hipMemcpy( d_target, target, n_target * sizeof( inode_t ), hipMemcpyHostToDevice ) );
  int ret =
    conn_->remoteConnect( i_source_host, d_source, n_source, i_target_host, d_target, n_target, conn_spec, syn_spec );
  CUDAFREECTRL( "d_source", d_source );
  CUDAFREECTRL( "d_target", d_target );

  return ret;
}

int
NESTGPU::RemoteConnect( int i_source_host,
  NodeSeq source,
  int i_target_host,
  NodeSeq target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  return RemoteConnect( i_source_host, source.i0, source.n, i_target_host, target.i0, target.n, conn_spec, syn_spec );
}

int
NESTGPU::RemoteConnect( int i_source_host,
  NodeSeq source,
  int i_target_host,
  std::vector< inode_t > target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  return RemoteConnect(
    i_source_host, source.i0, source.n, i_target_host, target.data(), target.size(), conn_spec, syn_spec );
}

int
NESTGPU::RemoteConnect( int i_source_host,
  std::vector< inode_t > source,
  int i_target_host,
  NodeSeq target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  return RemoteConnect(
    i_source_host, source.data(), source.size(), i_target_host, target.i0, target.n, conn_spec, syn_spec );
}

int
NESTGPU::RemoteConnect( int i_source_host,
  std::vector< inode_t > source,
  int i_target_host,
  std::vector< inode_t > target,
  ConnSpec& conn_spec,
  SynSpec& syn_spec )
{
  return RemoteConnect(
    i_source_host, source.data(), source.size(), i_target_host, target.data(), target.size(), conn_spec, syn_spec );
}
