#include "hip/hip_runtime.h"
#include <config.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "cuda_error.h"

unsigned int *curand_int(hiprandGenerator_t &gen, size_t n)
{
  unsigned int *dev_data;
  // Allocate n integers on host
  unsigned int *host_data = new unsigned int[n];

  // Allocate n integers on device
  CUDA_CALL(hipMalloc((void **)&dev_data, n*sizeof(unsigned int)));
  // Create pseudo-random number generator

  // Generate n integers on device
  CURAND_CALL(hiprandGenerate(gen, dev_data, n));
  hipDeviceSynchronize();
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(host_data, dev_data, n*sizeof(unsigned int),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(dev_data));

  return host_data;
}

float *hiprand_uniform(hiprandGenerator_t &gen, size_t n)
{
  float *dev_data;
  // Allocate n floats on host
  float *host_data = new float[n];

  // Allocate n floats on device
  CUDA_CALL(hipMalloc((void **)&dev_data, n*sizeof(float)));
  // Create pseudo-random number generator

  // Generate n integers on device
  CURAND_CALL(hiprandGenerateUniform(gen, dev_data, n));
  hipDeviceSynchronize();
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(host_data, dev_data, n*sizeof(float),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(dev_data));

  return host_data;
}

float *hiprand_normal(hiprandGenerator_t &gen, size_t n, float mean,
		     float stddev)
{
  size_t n1 = ( (n % 2) == 0 ) ? n : n + 1; // round up to multiple of 2
  float *dev_data;
  // Allocate n floats on host
  float *host_data = new float[n];

  // Allocate n1 floats on device
  CUDA_CALL(hipMalloc((void **)&dev_data, n1*sizeof(float)));
  // Create pseudo-random number generator

  // Generate n1 integers on device
  //printf("hiprandGenerateNormal n1: %d\tmean: %f\tstd: %f\n", (int)n1, mean,
  //	 stddev);
  CURAND_CALL(hiprandGenerateNormal(gen, dev_data, n1, mean, stddev));
  hipDeviceSynchronize();
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(host_data, dev_data, n*sizeof(float),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(dev_data));

  return host_data;
}
