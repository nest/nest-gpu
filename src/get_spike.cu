#include "hip/hip_runtime.h"
/*
 *  get_spike.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */





#include <config.h>
#include <stdio.h>

#include "nestgpu.h"
#include "node_group.h"
#include "send_spike.h"
#include "spike_buffer.h"
#include "cuda_error.h"

extern __constant__ long long NESTGPUTimeIdx;
extern __constant__ float NESTGPUTimeResolution;
extern __constant__ NodeGroupStruct NodeGroupArray[];
extern __device__ signed char *NodeGroupMap;

extern __device__ void SynapseUpdate(int syn_group, float *w, float Dt);

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

//////////////////////////////////////////////////////////////////////
// This is the function called by the nested loop
// that collects the spikes
__device__ void CollectSpikeFunction(int i_spike, int i_syn)
{
  int i_source = SpikeSourceIdx[i_spike];
  int i_conn = SpikeConnIdx[i_spike];
  float height = SpikeHeight[i_spike];
  unsigned int target_port
    = ConnectionGroupTargetNode[i_conn*NSpikeBuffer + i_source][i_syn];
  int i_target = target_port & PORT_MASK;
  unsigned char port = (unsigned char)(target_port >> (PORT_N_SHIFT + 24));
  unsigned char syn_group
    = ConnectionGroupTargetSynGroup[i_conn*NSpikeBuffer + i_source][i_syn];
  float weight = ConnectionGroupTargetWeight[i_conn*NSpikeBuffer+i_source]
    [i_syn];
  //printf("handles spike %d src %d conn %d syn %d target %d"
  //" port %d weight %f\n",
  //i_spike, i_source, i_conn, i_syn, i_target,
  //port, weight);

  /////////////////////////////////////////////////////////////////
  int i_group=NodeGroupMap[i_target];
  int i = port*NodeGroupArray[i_group].n_node_ + i_target
    - NodeGroupArray[i_group].i_node_0_;
  double d_val = (double)(height*weight);

  atomicAddDouble(&NodeGroupArray[i_group].get_spike_array_[i], d_val);
  if (syn_group>0) {
    ConnectionGroupTargetSpikeTime[i_conn*NSpikeBuffer+i_source][i_syn]
      = (unsigned short)(NESTGPUTimeIdx & 0xffff);

    long long Dt_int = NESTGPUTimeIdx - LastRevSpikeTimeIdx[i_target];
     if (Dt_int>0 && Dt_int<MAX_SYN_DT) {
       SynapseUpdate(syn_group, &ConnectionGroupTargetWeight
		    [i_conn*NSpikeBuffer+i_source][i_syn],
		     -NESTGPUTimeResolution*Dt_int);
    }
  }
  ////////////////////////////////////////////////////////////////
}

__global__ void CollectSpikeKernel(int n_spikes, int *SpikeTargetNum)
{
  const int i_spike = blockIdx.x;
  if (i_spike<n_spikes) {
    const int n_spike_targets = SpikeTargetNum[i_spike];
    for (int i_syn = threadIdx.x; i_syn < n_spike_targets; i_syn += blockDim.x){
      CollectSpikeFunction(i_spike, i_syn);
    }
  }
}



///////////////

// improve using a grid
/*
__global__ void GetSpikes(double *spike_array, int array_size, int n_port,
			  int n_var,
			  float *port_weight_arr,
			  int port_weight_arr_step,
			  int port_weight_port_step,
			  float *port_input_arr,
			  int port_input_arr_step,
			  int port_input_port_step)
{
  int i_array = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_array < array_size*n_port) {
     int i_target = i_array % array_size;
     int port = i_array / array_size;
     int port_input = i_target*port_input_arr_step
       + port_input_port_step*port;
     int port_weight = i_target*port_weight_arr_step
       + port_weight_port_step*port;
     double d_val = (double)port_input_arr[port_input]
       + spike_array[i_array]
       * port_weight_arr[port_weight];

     port_input_arr[port_input] = (float)d_val;
  }
}
*/

__global__ void GetSpikes(double *spike_array, int array_size, int n_port,
			  int n_var,
			  float *port_weight_arr,
			  int port_weight_arr_step,
			  int port_weight_port_step,
			  float *port_input_arr,
			  int port_input_arr_step,
			  int port_input_port_step)
{
  int i_target = blockIdx.x*blockDim.x+threadIdx.x;
  int port = blockIdx.y*blockDim.y+threadIdx.y;

  if (i_target < array_size && port<n_port) {
    int i_array = port*array_size + i_target;
    int port_input = i_target*port_input_arr_step
      + port_input_port_step*port;
    int port_weight = i_target*port_weight_arr_step
      + port_weight_port_step*port;
    double d_val = (double)port_input_arr[port_input]
      + spike_array[i_array]
      * port_weight_arr[port_weight];

    port_input_arr[port_input] = (float)d_val;
  }
}


int NESTGPU::ClearGetSpikeArrays()
{
  for (unsigned int i=0; i<node_vect_.size(); i++) {
    BaseNeuron *bn = node_vect_[i];
    if (bn->get_spike_array_ != NULL) {
      gpuErrchk(hipMemsetAsync(bn->get_spike_array_, 0, bn->n_node_*bn->n_port_
			   *sizeof(double)));
    }
  }

  return 0;
}

int NESTGPU::FreeGetSpikeArrays()
{
  for (unsigned int i=0; i<node_vect_.size(); i++) {
    BaseNeuron *bn = node_vect_[i];
    if (bn->get_spike_array_ != NULL) {
      gpuErrchk(hipFree(bn->get_spike_array_));
    }
  }

  return 0;
}
