/*
 *  aeif_psc_exp.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "aeif_psc_exp.h"
#include "aeif_psc_exp_kernel.h"
#include "rk5.h"
#include <cmath>
#include <config.h>
#include <iostream>

namespace aeif_psc_exp_ns
{

__device__ void
NodeInit( int n_var, int n_param, double x, float* y, float* param, aeif_psc_exp_rk5 data_struct )
{
  // int array_idx = threadIdx.x + blockIdx.x * blockDim.x;

  V_th = -50.4;
  Delta_T = 2.0;
  g_L = 30.0;
  E_L = -70.6;
  C_m = 281.0;
  a = 4.0;
  b = 80.5;
  tau_w = 144.0;
  I_e = 0.0;
  V_peak = 0.0;
  V_reset = -60.0;
  t_ref = 0.0;
  den_delay = 0.0;

  I_syn_ex = 0;
  I_syn_in = 0;
  V_m = E_L;
  w = 0;
  tau_syn_ex = 0.2;
  tau_syn_in = 2.0;
  refractory_step = 0;
}

__device__ void
NodeCalibrate( int n_var, int n_param, double x, float* y, float* param, aeif_psc_exp_rk5 data_struct )
{
  // int array_idx = threadIdx.x + blockIdx.x * blockDim.x;

  refractory_step = 0;
  // set the right threshold depending on Delta_T
  if ( Delta_T <= 0.0 )
  {
    V_peak = V_th; // same as IAF dynamics for spikes if Delta_T == 0.
  }
}

} // namespace aeif_psc_exp_ns

__device__ void
NodeInit( int n_var, int n_param, double x, float* y, float* param, aeif_psc_exp_rk5 data_struct )
{
  aeif_psc_exp_ns::NodeInit( n_var, n_param, x, y, param, data_struct );
}

__device__ void
NodeCalibrate( int n_var, int n_param, double x, float* y, float* param, aeif_psc_exp_rk5 data_struct )

{
  aeif_psc_exp_ns::NodeCalibrate( n_var, n_param, x, y, param, data_struct );
}

using namespace aeif_psc_exp_ns;

int
aeif_psc_exp::Init( int i_node_0, int n_node, int n_port, int i_group )
{
  BaseNeuron::Init( i_node_0, n_node, n_port, i_group );
  node_type_ = i_aeif_psc_exp_model;
  n_scal_var_ = N_SCAL_VAR;
  n_scal_param_ = N_SCAL_PARAM;
  n_group_param_ = N_GROUP_PARAM;

  n_var_ = n_scal_var_;
  n_param_ = n_scal_param_;

  group_param_ = new float[ N_GROUP_PARAM ];

  scal_var_name_ = aeif_psc_exp_scal_var_name;
  scal_param_name_ = aeif_psc_exp_scal_param_name;
  group_param_name_ = aeif_psc_exp_group_param_name;
  // rk5_data_struct_.node_type_ = i_aeif_psc_exp_model;
  rk5_data_struct_.i_node_0_ = i_node_0_;

  SetGroupParam( "h_min_rel", 1.0e-3 );
  SetGroupParam( "h0_rel", 1.0e-2 );
  h_ = h0_rel_ * 0.1;

  rk5_.Init( n_node, n_var_, n_param_, 0.0, h_, rk5_data_struct_ );
  var_arr_ = rk5_.GetYArr();
  param_arr_ = rk5_.GetParamArr();

  // multiplication factor of input signal is always 1 for all nodes
  float input_weight = 1.0;
  CUDAMALLOCCTRL( "&port_weight_arr_", &port_weight_arr_, sizeof( float ) );
  gpuErrchk( hipMemcpy( port_weight_arr_, &input_weight, sizeof( float ), hipMemcpyHostToDevice ) );
  port_weight_arr_step_ = 0;
  port_weight_port_step_ = 0;

  port_input_arr_ = GetVarArr() + GetScalVarIdx( "I_syn_ex" );
  port_input_arr_step_ = n_var_;
  port_input_port_step_ = 1;
  den_delay_arr_ = GetParamArr() + GetScalParamIdx( "den_delay" );

  return 0;
}

int
aeif_psc_exp::Calibrate( double time_min, float time_resolution )
{
  h_min_ = h_min_rel_ * time_resolution;
  h_ = h0_rel_ * time_resolution;
  rk5_.Calibrate( time_min, h_, rk5_data_struct_ );

  return 0;
}

int
aeif_psc_exp::Update( long long it, double t1 )
{
  rk5_.Update< N_SCAL_VAR, N_SCAL_PARAM >( t1, h_min_, rk5_data_struct_ );

  return 0;
}
