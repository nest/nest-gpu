#include "hip/hip_runtime.h"
/*
 *  aeif_cond_beta.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */


#include "aeif_cond_beta.h"
#include "aeif_cond_beta_kernel.h"
#include "rk5.h"
#include <cmath>
#include <config.h>
#include <iostream>

namespace aeif_cond_beta_ns
{

__device__ void
NodeInit( int n_var, int n_param, double x, float* y, float* param, aeif_cond_beta_rk5 data_struct )
{
  // int array_idx = threadIdx.x + blockIdx.x * blockDim.x;

  V_th = -50.4;
  Delta_T = 2.0;
  g_L = 30.0;
  E_L = -70.6;
  C_m = 281.0;
  a = 4.0;
  b = 80.5;
  tau_w = 144.0;
  I_e = 0.0;
  V_peak = 0.0;
  V_reset = -60.0;
  t_ref = 0.0;
  den_delay = 0.0;
  E_rev_ex = 0.0;
  E_rev_in = 0.0;
  tau_decay_ex = 20.0;
  tau_decay_in = 20.0;
  tau_rise_ex = 2.0;
  tau_rise_in = 2.0;

  V_m = E_L;
  w = 0;
  refractory_step = 0;
  g_ex = 0;
  g_in = 0;
  g1_ex = 0;
  g1_in = 0;
}

__device__ void
NodeCalibrate( int n_var, int n_param, double x, float* y, float* param, aeif_cond_beta_rk5 data_struct )
{
  // int array_idx = threadIdx.x + blockIdx.x * blockDim.x;

  refractory_step = 0;

  // denominator is computed here to check that it is != 0
  float denom1 = tau_decay_ex - tau_rise_ex;
  float denom2 = 0;
  if ( denom1 != 0 )
  {
    // peak time
    float t_p = tau_decay_ex * tau_rise_ex * log( tau_decay_ex / tau_rise_ex ) / denom1;
    // another denominator is computed here to check that it is != 0
    denom2 = exp( -t_p / tau_decay_ex ) - exp( -t_p / tau_rise_ex );
  }
  if ( denom2 == 0 )
  { // if rise time == decay time use alpha function
    // use normalization for alpha function in this case
    g0_ex = M_E / tau_decay_ex;
  }
  else
  { // if rise time != decay time use beta function
    // normalization factor for conductance
    g0_ex = ( 1. / tau_rise_ex - 1. / tau_decay_ex ) / denom2;
  }

  denom1 = tau_decay_in - tau_rise_in;
  denom2 = 0;
  if ( denom1 != 0 )
  {
    // peak time
    float t_p = tau_decay_in * tau_rise_in * log( tau_decay_in / tau_rise_in ) / denom1;
    // another denominator is computed here to check that it is != 0
    denom2 = exp( -t_p / tau_decay_in ) - exp( -t_p / tau_rise_in );
  }
  if ( denom2 == 0 )
  { // if rise time == decay time use alpha function
    // use normalization for alpha function in this case
    g0_in = M_E / tau_decay_in;
  }
  else
  { // if rise time != decay time use beta function
    // normalization factor for conductance
    g0_in = ( 1. / tau_rise_in - 1. / tau_decay_in ) / denom2;
  }
}
}

__device__ void
NodeInit( int n_var, int n_param, double x, float* y, float* param, aeif_cond_beta_rk5 data_struct )
{
  aeif_cond_beta_ns::NodeInit( n_var, n_param, x, y, param, data_struct );
}

__device__ void
NodeCalibrate( int n_var, int n_param, double x, float* y, float* param, aeif_cond_beta_rk5 data_struct )

{
  aeif_cond_beta_ns::NodeCalibrate( n_var, n_param, x, y, param, data_struct );
}

using namespace aeif_cond_beta_ns;

int
aeif_cond_beta::Init( int i_node_0, int n_node, int n_port, int i_group, unsigned long long* seed )
{
  BaseNeuron::Init( i_node_0, n_node, 2 /*n_port*/, i_group, seed );
  node_type_ = i_aeif_cond_beta_model;
  n_scal_var_ = N_SCAL_VAR;
  n_scal_param_ = N_SCAL_PARAM;
  n_group_param_ = N_GROUP_PARAM;

  n_var_ = n_scal_var_;
  n_param_ = n_scal_param_;

  group_param_ = new float[ N_GROUP_PARAM ];

  scal_var_name_ = aeif_cond_beta_scal_var_name;
  scal_param_name_ = aeif_cond_beta_scal_param_name;
  group_param_name_ = aeif_cond_beta_group_param_name;
  // rk5_data_struct_.node_type_ = i_aeif_cond_beta_model;
  rk5_data_struct_.i_node_0_ = i_node_0_;

  SetGroupParam( "h_min_rel", 1.0e-3 );
  SetGroupParam( "h0_rel", 1.0e-2 );
  h_ = h0_rel_ * 0.1;

  rk5_.Init( n_node, n_var_, n_param_, 0.0, h_, rk5_data_struct_ );
  var_arr_ = rk5_.GetYArr();
  param_arr_ = rk5_.GetParamArr();

  port_weight_arr_ = GetParamArr() + GetScalParamIdx( "g0_ex" );
  port_weight_arr_step_ = n_param_;
  port_weight_port_step_ = 1;

  port_input_arr_ = GetVarArr() + GetScalVarIdx( "g1_ex" );
  port_input_arr_step_ = n_var_;
  port_input_port_step_ = 1;
  den_delay_arr_ = GetParamArr() + GetScalParamIdx( "den_delay" );

  return 0;
}

int
aeif_cond_beta::Calibrate( double time_min, float time_resolution )
{
  h_min_ = h_min_rel_ * time_resolution;
  h_ = h0_rel_ * time_resolution;
  rk5_.Calibrate( time_min, h_, rk5_data_struct_ );

  return 0;
}

int
aeif_cond_beta::Update( long long it, double t1 )
{
  rk5_.Update< N_SCAL_VAR, N_SCAL_PARAM >( t1, h_min_, rk5_data_struct_ );

  return 0;
}
