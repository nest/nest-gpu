#include "hip/hip_runtime.h"
/*
 *  syn_model.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */





#include <config.h>
#include <iostream>
#include "ngpu_exception.h"
#include "cuda_error.h"
#include "nestgpu.h"
#include "syn_model.h"
#include "test_syn_model.h"
#include "stdp.h"

int *d_SynGroupTypeMap;
__device__ int *SynGroupTypeMap;

float **d_SynGroupParamMap;
__device__ float **SynGroupParamMap;

__device__ void TestSynModelUpdate(float *w, float Dt, float *param);

__device__ void STDPUpdate(float *w, float Dt, float *param);

__device__ void SynapseUpdate(int syn_group, float *w, float Dt)
{
  int syn_type = SynGroupTypeMap[syn_group-1];
  float *param = SynGroupParamMap[syn_group-1];
  switch(syn_type) {
  case i_test_syn_model:
    TestSynModelUpdate(w, Dt, param);
    break;
  case i_stdp_model:
    STDPUpdate(w, Dt, param);
    break;
  }
}


__global__ void SynGroupInit(int *syn_group_type_map,
			     float **syn_group_param_map)
{
  SynGroupTypeMap = syn_group_type_map;
  SynGroupParamMap = syn_group_param_map;

}

int SynModel::GetNParam()
{
  return n_param_;
}

std::vector<std::string> SynModel::GetParamNames()
{
  std::vector<std::string> param_name_vect;
  for (int i=0; i<n_param_; i++) {
    param_name_vect.push_back(param_name_[i]);
  }

  return param_name_vect;
}

bool SynModel::IsParam(std::string param_name)
{
  int i_param;
  for (i_param=0; i_param<n_param_; i_param++) {
    if (param_name == param_name_[i_param]) return true;
  }
  return false;
}

int SynModel::GetParamIdx(std::string param_name)
{
  int i_param;
  for (i_param=0; i_param<n_param_; i_param++) {
    if (param_name == param_name_[i_param]) break;
  }
  if (i_param == n_param_) {
    throw ngpu_exception(std::string("Unrecognized parameter ")
			 + param_name);
  }

  return i_param;
}

float SynModel::GetParam(std::string param_name)
{
  if (!IsParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized synapse parameter ")
			 + param_name);
  }
  int i_param =  GetParamIdx(param_name);
  float *d_param_pt = d_param_arr_ + i_param;
  float param_val;
  gpuErrchk(hipMemcpy(&param_val, d_param_pt, sizeof(float),
		       hipMemcpyDeviceToHost));
  return param_val;
}


int SynModel::SetParam(std::string param_name, float val)
{
  if (!IsParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized synapse parameter ")
			 + param_name);
  }
  int i_param =  GetParamIdx(param_name);
  float *d_param_pt = d_param_arr_ + i_param;
  gpuErrchk(hipMemcpy(d_param_pt, &val, sizeof(float),
		       hipMemcpyHostToDevice));
  return 0;
}


int NESTGPU::CreateSynGroup(std::string model_name)
{
  CheckUncalibrated("Nodes cannot be created after calibration");
  if (model_name == syn_model_name[i_test_syn_model]) {
    TestSynModel *test_syn_model_group = new TestSynModel;
    syn_group_vect_.push_back(test_syn_model_group);
  }
  else if (model_name == syn_model_name[i_stdp_model]) {
    STDP *stdp_group = new STDP;
    syn_group_vect_.push_back(stdp_group);
  }
  else {
    throw ngpu_exception(std::string("Unknown synapse model name: ")
			 + model_name);
  }
  return syn_group_vect_.size(); // 0 is standard synapse
}

int NESTGPU::GetSynGroupNParam(int syn_group)
{
  if (syn_group<1 || syn_group>(int)syn_group_vect_.size()) {
    throw ngpu_exception("Unrecognized synapse group");
  }

  return syn_group_vect_[syn_group-1]->GetNParam();
}

std::vector<std::string> NESTGPU::GetSynGroupParamNames(int syn_group)
{
  if (syn_group<1 || syn_group>(int)syn_group_vect_.size()) {
    throw ngpu_exception("Unrecognized synapse group");
  }

  return syn_group_vect_[syn_group-1]->GetParamNames();
}

bool NESTGPU::IsSynGroupParam(int syn_group, std::string param_name)
{
  if (syn_group<1 || syn_group>(int)syn_group_vect_.size()) {
    throw ngpu_exception("Unrecognized synapse group");
  }

  return syn_group_vect_[syn_group-1]->IsParam(param_name);
}

int NESTGPU::GetSynGroupParamIdx(int syn_group, std::string param_name)
{
  if (syn_group<1 || syn_group>(int)syn_group_vect_.size()) {
    throw ngpu_exception("Unrecognized synapse group");
  }

  return syn_group_vect_[syn_group-1]->GetParamIdx(param_name);
}

float NESTGPU::GetSynGroupParam(int syn_group, std::string param_name)
{
  if (syn_group<1 || syn_group>(int)syn_group_vect_.size()) {
    throw ngpu_exception("Unrecognized synapse group");
  }

  return syn_group_vect_[syn_group-1]->GetParam(param_name);
}

int NESTGPU::SetSynGroupParam(int syn_group, std::string param_name,
				float val)
{
  if (syn_group<1 || syn_group>(int)syn_group_vect_.size()) {
    throw ngpu_exception("Unrecognized synapse group");
  }

  return syn_group_vect_[syn_group-1]->SetParam(param_name, val);
}


int NESTGPU::SynGroupCalibrate()
{
  int n_group = syn_group_vect_.size();
  int *h_SynGroupTypeMap = new int[n_group];
  float **h_SynGroupParamMap = new float*[n_group];

  for (int syn_group=1; syn_group<=n_group; syn_group++) {
    h_SynGroupTypeMap[syn_group-1] = syn_group_vect_[syn_group-1]->type_;
    h_SynGroupParamMap[syn_group-1]
      = syn_group_vect_[syn_group-1]->d_param_arr_;
  }
  gpuErrchk(hipMalloc(&d_SynGroupTypeMap, n_group*sizeof(int)));
  gpuErrchk(hipMalloc(&d_SynGroupParamMap, n_group*sizeof(float*)));

  // Memcopies will be synchronised with SynGroupInit kernel
  gpuErrchk(hipMemcpyAsync(d_SynGroupTypeMap, h_SynGroupTypeMap,
		       n_group*sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyAsync(d_SynGroupParamMap, h_SynGroupParamMap,
		       n_group*sizeof(float*), hipMemcpyHostToDevice));

  SynGroupInit<<<1,1>>>(d_SynGroupTypeMap, d_SynGroupParamMap);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  delete[] h_SynGroupTypeMap;
  delete[] h_SynGroupParamMap;

  return 0;
}
