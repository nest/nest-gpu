#include "hip/hip_runtime.h"
/*
 *  poisson.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */


#include "cuda_error.h"
#include "poisson.h"
#include "spike_buffer.h"
#include <algorithm>
#include <config.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>

__device__ unsigned int* PoissonData;

__global__ void
PoissonUpdate( unsigned int* poisson_data )
{
  PoissonData = poisson_data;
}

__global__ void
PoissonSendSpikes( int i_node_0, int n_node )
{
  int i_node = threadIdx.x + blockIdx.x * blockDim.x;
  if ( i_node < n_node )
  {
    int i_node_abs = i_node_0 + i_node;
    unsigned int height = PoissonData[ i_node ];
    if ( height > 0 )
    {
      PushSpike( i_node_abs, ( float ) height );
    }
  }
}

__global__ void
FixPoissonGenerator( unsigned int* poisson_data, int n, float mean )
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if ( i < n )
  {
    unsigned int val = poisson_data[ i ];
    if ( val > mean * 5 )
    {
      poisson_data[ i ] = 0;
    }
  }
}

int
PoissonGenerator::Init( hiprandGenerator_t* random_generator, unsigned int n )
{
  poisson_data_size_ = n;
  // Allocate n integers on device
  CUDA_CALL( hipMalloc( ( void** ) &dev_poisson_data_, n * sizeof( unsigned int ) ) );
  random_generator_ = random_generator;

  return 0;
}

int
PoissonGenerator::Generate()
{
  return Generate( n_steps_ );
}

int
PoissonGenerator::Generate( int max_n_steps )
{
  if ( max_n_steps <= 0 )
  {
    more_steps_ = n_steps_;
  }
  else
  {
    more_steps_ = std::min( n_steps_, max_n_steps );
  }
  // Generate N floats on device
  CURAND_CALL( hiprandGeneratePoisson( *random_generator_, dev_poisson_data_, n_node_ * more_steps_, lambda_ ) );
  FixPoissonGenerator<<< ( n_node_ + 1023 ) / 1024, 1024 >>>( dev_poisson_data_, n_node_ * more_steps_, lambda_ );

  return 0;
}

int
PoissonGenerator::Free()
{
  CUDA_CALL( hipFree( dev_poisson_data_ ) );

  return 0;
}

PoissonGenerator::~PoissonGenerator()
{
  // Free();
}

PoissonGenerator::PoissonGenerator()
{
  buffer_size_ = 100000;
  n_node_ = 0;
}

int
PoissonGenerator::Create( hiprandGenerator_t* random_generator, int i_node_0, int n_node, float lambda )
{
  i_node_0_ = i_node_0;
  n_node_ = n_node;
  lambda_ = lambda;

  n_steps_ = ( buffer_size_ - 1 ) / n_node + 1;
  // with the above formula:
  // buffer_size <= n_node*n_steps <= buffer_size + n_node - 1
  Init( random_generator, n_node_ * n_steps_ );
  i_step_ = 0;

  return 0;
}

int
PoissonGenerator::Update( int max_n_steps )
{
  if ( i_step_ == 0 )
  {
    Generate( max_n_steps );
  }

  if ( i_step_ == more_steps_ )
  {
    throw ngpu_exception(
      "Step index larger than maximum number of steps "
      "in poisson generator" );
  }

  PoissonUpdate<<< 1, 1 >>>( &dev_poisson_data_[ i_step_ * n_node_ ] );
  gpuErrchk( hipPeekAtLastError() );

  PoissonSendSpikes<<< ( n_node_ + 1023 ) / 1024, 1024 >>>( i_node_0_, n_node_ );
  gpuErrchk( hipPeekAtLastError() );

  i_step_++;
  if ( i_step_ == n_steps_ )
  {
    i_step_ = 0;
  }

  return 0;
}
