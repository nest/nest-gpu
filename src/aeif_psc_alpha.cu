/*
 *  aeif_psc_alpha.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "aeif_psc_alpha.h"
#include "aeif_psc_alpha_kernel.h"
#include "rk5.h"
#include <cmath>
#include <config.h>
#include <iostream>

namespace aeif_psc_alpha_ns
{

__device__ void
NodeInit( int n_var, int n_param, double x, float* y, float* param, aeif_psc_alpha_rk5 data_struct )
{
  // int array_idx = threadIdx.x + blockIdx.x * blockDim.x;

  V_th = -50.4;
  Delta_T = 2.0;
  g_L = 30.0;
  E_L = -70.6;
  C_m = 281.0;
  a = 4.0;
  b = 80.5;
  tau_w = 144.0;
  I_e = 0.0;
  V_peak = 0.0;
  V_reset = -60.0;
  t_ref = 0.0;
  den_delay = 0.0;

  V_m = E_L;
  w = 0.0;
  refractory_step = 0;
  I_syn_ex = 0.0;
  I_syn_in = 0.0;
  I1_syn_ex = 0.0;
  I1_syn_in = 0.0;
  tau_syn_ex = 0.2;
  tau_syn_in = 2.0;
}

__device__ void
NodeCalibrate( int n_var, int n_param, double x, float* y, float* param, aeif_psc_alpha_rk5 data_struct )
{
  // int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
  refractory_step = 0;
  // set the right threshold depending on Delta_T
  if ( Delta_T <= 0.0 )
  {
    V_peak = V_th; // same as IAF dynamics for spikes if Delta_T == 0.
  }
  I0_ex = M_E / tau_syn_ex;
  I0_in = M_E / tau_syn_in;
}

} // namespace aeif_psc_alpha_ns

__device__ void
NodeInit( int n_var, int n_param, double x, float* y, float* param, aeif_psc_alpha_rk5 data_struct )
{
  aeif_psc_alpha_ns::NodeInit( n_var, n_param, x, y, param, data_struct );
}

__device__ void
NodeCalibrate( int n_var, int n_param, double x, float* y, float* param, aeif_psc_alpha_rk5 data_struct )

{
  aeif_psc_alpha_ns::NodeCalibrate( n_var, n_param, x, y, param, data_struct );
}

using namespace aeif_psc_alpha_ns;

int
aeif_psc_alpha::Init( int i_node_0, int n_node, int n_port, int i_group )
{
  BaseNeuron::Init( i_node_0, n_node, n_port, i_group );
  node_type_ = i_aeif_psc_alpha_model;
  n_scal_var_ = N_SCAL_VAR;
  n_scal_param_ = N_SCAL_PARAM;
  n_group_param_ = N_GROUP_PARAM;

  n_var_ = n_scal_var_;
  n_param_ = n_scal_param_;

  group_param_ = new float[ N_GROUP_PARAM ];

  scal_var_name_ = aeif_psc_alpha_scal_var_name;
  scal_param_name_ = aeif_psc_alpha_scal_param_name;
  group_param_name_ = aeif_psc_alpha_group_param_name;
  // rk5_data_struct_.node_type_ = i_aeif_psc_alpha_model;
  rk5_data_struct_.i_node_0_ = i_node_0_;

  SetGroupParam( "h_min_rel", 1.0e-3 );
  SetGroupParam( "h0_rel", 1.0e-2 );
  h_ = h0_rel_ * 0.1;

  rk5_.Init( n_node, n_var_, n_param_, 0.0, h_, rk5_data_struct_ );
  var_arr_ = rk5_.GetYArr();
  param_arr_ = rk5_.GetParamArr();

  port_weight_arr_ = GetParamArr() + GetScalParamIdx( "I0_ex" );
  port_weight_arr_step_ = n_param_;
  port_weight_port_step_ = 1;

  port_input_arr_ = GetVarArr() + GetScalVarIdx( "I1_syn_ex" );
  port_input_arr_step_ = n_var_;
  port_input_port_step_ = 1;
  den_delay_arr_ = GetParamArr() + GetScalParamIdx( "den_delay" );

  return 0;
}

int
aeif_psc_alpha::Calibrate( double time_min, float time_resolution )
{
  h_min_ = h_min_rel_ * time_resolution;
  h_ = h0_rel_ * time_resolution;
  rk5_.Calibrate( time_min, h_, rk5_data_struct_ );

  return 0;
}

int
aeif_psc_alpha::Update( long long it, double t1 )
{
  rk5_.Update< N_SCAL_VAR, N_SCAL_PARAM >( t1, h_min_, rk5_data_struct_ );

  return 0;
}
